
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu() { printf("Hello from the GPU!\n"); }

/*
 * @brief CUDA kernel for vector addition
 *
 * This kernel function computes the addition of a vector element
 *
 * @param A_d The first vector
 * @param B_d The second vector
 * @param C_d The result vector
 * */
__global__ void vec_add_kernel(float *d_A, float *d_B, float *d_C, int n) {

  // NOTE: The `__global__` keyword indicates that the function is a kernel and
  // that it can be called to generate a grid of threads on a device.

  // NOTE: a unique global index i is calculated
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // WARN: This is because not all vector lengths can be expressed as multiples
  // of the block size. For example, let’s assume that the vector length is 100,
  // the smallest efficient thread block dimension is 32. Assume that we picked
  // 32 as block size. One would need to launch 4 thread blocks to process all
  // the 100 vector elements. However, the 4 thread blocks would have 128
  // threads. We need to disable the last 28 threads in thread block 3 from
  // doing work not expected by the original program.
  // Since all threads are to execute the same code, all will test their i
  // values against n, which is 100. With the if (i<n) statement, the first 100
  // threads will perform the addition, whereas the last 28 will not. This
  // allows the kernel to be called to process vectors of arbitrary lengths.
  if (i < n) {

    // NOTE: Note that all the thread blocks operate on different parts of the
    // vectors. They can be executed in any arbitrary order. The programmer must
    // not make any assumptions regarding execution order.
    d_C[i] = d_A[i] + d_B[i];
  }
}

/**
 * @brief Sequential vector addition
 *
 * This functions adds two vector in a sequential fashion.
 *
 * @param h_A The first vector
 * @param h_B The second vector
 * @param h_C The result vector
 * @param n The length of the vectors
 * */

void seq_vec_add(float *h_A, float *h_B, float *h_C, int n) {
  for (int i = 0; i < n; i++) {
    h_C[i] = h_A[i] + h_B[i];
  }
}

/*
 * @brief Parallel vector addition
 *
 * This functions adds two vector in a parallel fashion.
 *
 * @param h_A The first vector
 * @param h_B The second vector
 * @param h_C The result vector
 * @param n The length of the vectors
 * */
void par_vecAdd(float *h_A, float *h_B, float *h_C, int n) {
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;

  // WARN: The first parameter to the cudaMalloc function is the address of a
  // pointer variable that will be set to point to the allocated object. The
  // address of the pointer variable should be cast to (void **) because the
  // function expects a generic pointer; the memory allocation function is a
  // generic function that is not restricted to any particular type of objects.
  //
  // NOTE: This parameter allows the cudaMalloc function to write the address of
  // the allocated memory into the provided pointer variable regardless of its
  // type. The host code that calls kernels passes this pointer value to the
  // kernels that need to access the allocated memory object.
  hipError_t errA = hipMalloc((void **)&d_A, size);
  if (errA != hipSuccess) {
    printf("%s in %s at line % d\n", hipGetErrorString(errA), __FILE__,
           __LINE__);
    exit(EXIT_FAILURE);
  }

  // WARN: CUDA API functions return flags that indicate whether an error has
  // occurred when they served the request. Most errors are due to inappropriate
  // argument values used in the call.
  hipError_t errB = hipMalloc((void **)&d_B, size);
  if (errA != hipSuccess) {
    printf("%s in %s at line % d\n", hipGetErrorString(errB), __FILE__,
           __LINE__);
    exit(EXIT_FAILURE);
  }

  hipError_t errC = hipMalloc((void **)&d_C, size);
  if (errA != hipSuccess) {
    printf("%s in %s at line % d\n", hipGetErrorString(errC), __FILE__,
           __LINE__);
    exit(EXIT_FAILURE);
  }

  // NOTE: The cudaMemcpy function takes four parameters. The first parameter is
  // a pointer to the destination location for the data object to be copied. The
  // second parameter points to the source location. The third parameter
  // specifies the number of bytes to be copied. The fourth parameter indicates
  // the types of memory involved in the copy: from host to host, from host to
  // device, from device to host, and from device to device.
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // NOTE: When the host code calls a kernel, it sets the grid and thread block
  // dimensions via execution configuration parameters.
  // The first configuration parameter gives the number of blocks in the grid.
  // The second specifies the number of threads in each block.
  vec_add_kernel<<<ceil(n / 256.0), 256>>>(d_A, d_B, d_C, n);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // NOTE: cudaFree does not need to change the value it only needs to use the
  // value of A_d to return the allocated memory back to the available pool.
  // Thus only the value and not the address of A_d is passed as an argument.
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int main() {
  printf("vector addition - CUDA implementation\n");
  printf("Chapter 2: Vector addition\n\n");

  // TODO: Implement test/demo code
  return 0;
}
